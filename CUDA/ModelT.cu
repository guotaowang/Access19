#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include"E:\Program Files\MATLAB\R2012b\extern\include\mex.h"
#include <stdio.h>
#include <algorithm>


__global__ void Transfer_kernel(double *Result, double* Result2,double * Result3,double * Result4,double *ModelNum,double* ModelSm,double *SpNum,double *ColorD,double *SumG,double *MaxDim,double* MainColor)
{
	int z = threadIdx.x;
	int MNum = (int)(*ModelNum), SNum = (int)(*SpNum),MDim=(int)(*MaxDim);
	if (threadIdx.x >= SNum)
		return;

	for (int k = 0; k < MNum; k++)
	{
		if (ModelSm[k] != 0&&ColorD[z+MDim*k]!=1000)
		{
			Result[z] = Result[z] + ModelSm[k] *exp(-ColorD[z+MDim*k])*SumG[z+MDim*k];//
            Result2[z+MDim*k]=exp(-MainColor[z+MDim*k]);
            Result4[z+MDim*k]=ModelSm[k] *exp(-ColorD[z+MDim*k])*SumG[z+MDim*k];
            Result3[z+MDim*k]=ModelSm[k] *exp(-ColorD[z+MDim*k])*SumG[z+MDim*k];
		}
	}
	return;

}
void ModelT(double *Result, double* Result2,double * Result3,double * Result4,double *ModelNum,double* ModelSm,double *SpNum,double *MaxDim,double *ColorD,double *SumG,double* MainColor)
{
	double * dev_Result,*dev_Result2,*dev_Result3,*dev_Result4;
	double *dev_ModelNum;
	double *dev_ModelSm;
	double *dev_SpNum,*dev_MaxDim;
    double *dev_ColorD, *dev_SumG,*dev_MainColor;
	int MDim = (int)(*MaxDim);
	int MNum = (int)(*ModelNum);
	int Spnum = (int)(*SpNum);

	hipMalloc((void **)&dev_Result, sizeof(double)* MDim);
    hipMalloc((void **)&dev_Result2, sizeof(double)* MDim*MNum);
    hipMalloc((void **)&dev_Result3, sizeof(double)* MDim*MNum);
    hipMalloc((void **)&dev_Result4, sizeof(double)* MDim*MNum);
	hipMalloc((void **)&dev_ModelSm, sizeof(double)* MNum);
	hipMalloc((void **)&dev_SpNum, sizeof(double));
    hipMalloc((void **)&dev_MaxDim, sizeof(double));
	hipMalloc((void **)&dev_ModelNum, sizeof(double));
    hipMalloc((void **)&dev_ColorD, sizeof(double)* MDim*MNum);
	hipMalloc((void **)&dev_SumG, sizeof(double)* MDim*MNum);
    hipMalloc((void **)&dev_MainColor, sizeof(double)* MDim*MNum);

	hipMemcpy(dev_ModelNum, ModelNum, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_SpNum, SpNum, sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_MaxDim, MaxDim, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_ModelSm, ModelSm, sizeof(double)* MNum, hipMemcpyHostToDevice);

    hipMemcpy(dev_ColorD, ColorD, sizeof(double)* MDim*MNum, hipMemcpyHostToDevice);
	hipMemcpy(dev_SumG, SumG, sizeof(double)* MDim*MNum, hipMemcpyHostToDevice);
    hipMemcpy(dev_MainColor, MainColor, sizeof(double)* MDim*MNum, hipMemcpyHostToDevice);

	dim3 threads(Spnum);
	Transfer_kernel << <1, threads >> >(dev_Result, dev_Result2, dev_Result3,dev_Result4, dev_ModelNum, dev_ModelSm, dev_SpNum, dev_ColorD,dev_SumG,dev_MaxDim,dev_MainColor);
	hipMemcpy(Result, dev_Result, sizeof(double)*MDim, hipMemcpyDeviceToHost);
    hipMemcpy(Result2, dev_Result2, sizeof(double)* MDim*MNum, hipMemcpyDeviceToHost);
    hipMemcpy(Result3, dev_Result3, sizeof(double)* MDim*MNum, hipMemcpyDeviceToHost);
    hipMemcpy(Result4, dev_Result4, sizeof(double)* MDim*MNum, hipMemcpyDeviceToHost);
	hipFree(dev_ModelSm);
	hipFree(dev_ModelNum);
	hipFree(dev_Result);
	hipFree(dev_Result2);
	hipFree(dev_Result3);
    hipFree(dev_Result4);
	hipFree(dev_SpNum);
	hipFree(dev_Result);
    hipFree(dev_ColorD);
	hipFree(dev_SumG);
    hipFree(dev_MainColor);

}




