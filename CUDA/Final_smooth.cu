#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include"E:\Program Files\MATLAB\R2012b\extern\include\mex.h"
#include <stdio.h>
#include <algorithm>


__global__ void smooth_kernel(double *FinS,double* midP,double * ring,double *MSaliencyM ,double *K1,double *N1,double *Par,double *spnum ,double *Par1,double *Par2)
{
	int i=blockIdx.x;
	int j=threadIdx.x;
    int K=(int)(*K1),N=(int)(*N1);
	if (blockIdx.x >= N || threadIdx.x >= spnum[i])
		return;

	double Lcolor1,Lcolor2,Lcolor3,Llocation1,Llocation2,Lcolor4,Lcolor5;
	double Rcolor1,Rcolor2,Rcolor3,Rlocation1,Rlocation2,Rcolor4,Rcolor5;
	double Tweight=0,weight1=0;
	double MSS=0;
	double CDist=0,LDist=0;
    Llocation1 = midP[(K)*11*i+j],Llocation2 = midP[(K)*11*i+(K)*1+j],Lcolor1 = midP[(K)*11*i+(K)*2+j], Lcolor2 = midP[(K)*11*i+(K)*3+j],Lcolor3 = midP[(K)*11*i+(K)*4+j],Lcolor4 = midP[(K)*11*i+(K)*9+j],Lcolor5 = midP[(K)*11*i+(K)*10+j];

	for (int k=0;k<spnum[i];k++)
	{
		Rlocation1 = midP[(K)*11*i+k],Rlocation2 = midP[(K)*11*i+(K)*1+k],Rcolor1 = midP[(K)*11*i+(K)*2+k], Rcolor2 = midP[(K)*11*i+(K)*3+k],Rcolor3 = midP[(K)*11*i+(K)*4+k],Rcolor4 = midP[(K)*11*i+(K)*9+k],Rcolor5 = midP[(K)*11*i+(K)*10+k];
		LDist=abs(Llocation1-Rlocation1)+abs(Llocation2-Rlocation2);
		CDist=sqrt((Lcolor1-Rcolor1)*(Lcolor1-Rcolor1)+(Lcolor3-Rcolor3)*(Lcolor3-Rcolor3)+(Lcolor2-Rcolor2)*(Lcolor2-Rcolor2)+(Lcolor4-Rcolor4)*(Lcolor4-Rcolor4)+(Lcolor5-Rcolor5)*(Lcolor5-Rcolor5));
       // CDist=(abs(Lcolor1-Rcolor1)+abs(Lcolor3-Rcolor3)+abs(Lcolor2-Rcolor2)+abs(Lcolor4-Rcolor4)+abs(Lcolor5-Rcolor5));
		if (LDist<min(max(ring[i*K+k],*Par1),*Par2))
		{
			weight1=exp(-CDist*(*Par));
			Tweight+=weight1;
			MSS=MSS+MSaliencyM[K*i+k]*weight1;
		}
	}
    if(i<N-1)
    for (int k=0;k<spnum[i+1];k++)
	{
		Rlocation1 = midP[(K)*11*(i+1)+k],Rlocation2 = midP[(K)*11*(i+1)+(K)*1+k],Rcolor1 = midP[(K)*11*(i+1)+(K)*2+k], Rcolor2 = midP[(K)*11*(i+1)+(K)*3+k],Rcolor3 = midP[(K)*11*(i+1)+(K)*4+k],Rcolor4 = midP[(K)*11*(i+1)+(K)*9+k],Rcolor5 = midP[(K)*11*(i+1)+(K)*10+k];
		LDist=abs(Llocation1-Rlocation1)+abs(Llocation2-Rlocation2);
        //CDist=(abs(Lcolor1-Rcolor1)+abs(Lcolor3-Rcolor3)+abs(Lcolor2-Rcolor2)+abs(Lcolor4-Rcolor4)+abs(Lcolor5-Rcolor5));
		CDist=sqrt((Lcolor1-Rcolor1)*(Lcolor1-Rcolor1)+(Lcolor3-Rcolor3)*(Lcolor3-Rcolor3)+(Lcolor2-Rcolor2)*(Lcolor2-Rcolor2)+(Lcolor4-Rcolor4)*(Lcolor4-Rcolor4)+(Lcolor5-Rcolor5)*(Lcolor5-Rcolor5));
		if (LDist<min(max(ring[(i+1)*K+k],*Par1),*Par2))
		{
			weight1=exp(-CDist*(*Par));
			Tweight+=weight1;
			MSS=MSS+MSaliencyM[K*(i+1)+k]*weight1;
		}
	}
    if (i>0)
    for (int k=0;k<spnum[i-1];k++)
	{
		Rlocation1 = midP[(K)*11*(i-1)+k],Rlocation2 = midP[(K)*11*(i-1)+(K)*1+k],Rcolor1 = midP[(K)*11*(i-1)+(K)*2+k], Rcolor2 = midP[(K)*11*(i-1)+(K)*3+k],Rcolor3 = midP[(K)*11*(i-1)+(K)*4+k],Rcolor4 = midP[(K)*11*(i-1)+(K)*9+k],Rcolor5 = midP[(K)*11*(i-1)+(K)*10+k];
		LDist=abs(Llocation1-Rlocation1)+abs(Llocation2-Rlocation2);
		CDist=sqrt((Lcolor1-Rcolor1)*(Lcolor1-Rcolor1)+(Lcolor3-Rcolor3)*(Lcolor3-Rcolor3)+(Lcolor2-Rcolor2)*(Lcolor2-Rcolor2)+(Lcolor4-Rcolor4)*(Lcolor4-Rcolor4)+(Lcolor5-Rcolor5)*(Lcolor5-Rcolor5));
        //CDist=(abs(Lcolor1-Rcolor1)+abs(Lcolor3-Rcolor3)+abs(Lcolor2-Rcolor2)+abs(Lcolor4-Rcolor4)+abs(Lcolor5-Rcolor5));
		if (LDist<min(max(ring[(i-1)*K+k],*Par1),*Par2))
		{
			weight1=exp(-CDist*(*Par));
			Tweight+=weight1;
			MSS=MSS+MSaliencyM[K*(i-1)+k]*weight1;
		}
	}
	MSS=MSS/Tweight;
	FinS[K*i+j]=MSS;
	return;

}
void Final_smooth(double *FinS,double* midP,double * ring,double *MSaliencyM,double *K1,double *N1,double *Par,double *spnum,double *Par1,double *Par2)
{
	double * dev_FinS;
	double *dev_mid,*dev_ring,*dev_MSaliencyM;
    double *dev_K1,*dev_N1, *dev_Par,*dev_spnum,*dev_Par1,*dev_Par2;
    int K=(int)(*K1),N=(int)(*N1);

	hipMalloc((void **)&dev_mid, sizeof(double)* (K) * 11 * N);
	hipMalloc((void **)&dev_ring, sizeof(double)* K * N);
	hipMalloc((void **)&dev_MSaliencyM, sizeof(double)* K * N);
	hipMalloc((void **)&dev_FinS, sizeof(double)* K * N);
    hipMalloc((void **)&dev_K1, sizeof(double));
    hipMalloc((void **)&dev_N1, sizeof(double));
    hipMalloc((void **)&dev_Par, sizeof(double));
    hipMalloc((void **)&dev_Par1, sizeof(double));
    hipMalloc((void **)&dev_Par2, sizeof(double));
    hipMalloc((void **)&dev_spnum, sizeof(double)*N);

	hipMemcpy(dev_K1, K1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_N1, N1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_mid, midP, sizeof(double)* (K) * 11 * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_ring, ring, sizeof(double)* K*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_MSaliencyM, MSaliencyM, sizeof(double)* K*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_Par, Par, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_Par1, Par1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_Par2, Par2, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_spnum, spnum, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 threads(K);
	dim3 grids(N);
	smooth_kernel << <grids, threads >> >(dev_FinS,dev_mid,dev_ring,dev_MSaliencyM,dev_K1,dev_N1,dev_Par,dev_spnum,dev_Par1,dev_Par2);

	hipMemcpy(FinS, dev_FinS, sizeof(double)*K*N, hipMemcpyDeviceToHost);

	hipFree(dev_mid);
	hipFree(dev_FinS);
	hipFree(dev_MSaliencyM);
    hipFree(dev_ring);
    hipFree(dev_K1);
    hipFree(dev_N1);
    hipFree(dev_Par);
    hipFree(dev_spnum);
    hipFree(dev_Par1);
    hipFree(dev_Par2);
}	




