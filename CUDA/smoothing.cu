#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include"E:\Program Files\MATLAB\R2012b\extern\include\mex.h"
#include <stdio.h>
#include <algorithm>


__global__ void smooth_kernel(double *FinS, double * FinC,double* midP,double * ring,double *MSaliencyM,double* CSaliencyM ,double *K1,double *N1,double *Par,double *spnum ,double *Par1)
{
	int i=blockIdx.x;
	int j=threadIdx.x;
    int K=(int)(*K1),N=(int)(*N1);
	if (blockIdx.x >= N || threadIdx.x >= spnum[i])
		return;

	double Lcolor1,Lcolor2,Lcolor3,Llocation1,Llocation2;
	double Rcolor1,Rcolor2,Rcolor3,Rlocation1,Rlocation2;
    double Rcolor21,Rcolor22,Rcolor23,Rcolor31,Rcolor32,Rcolor33;
    double CDist2=0,CDist3=0;
	double Tweight=0,weight1=0,weight2=0,weight3=0;
	double MSS=0,CSS=0;
	double CDist=0,LDist=0;
    Llocation1 = midP[(K)*5*i+j],Llocation2 = midP[(K)*5*i+(K)*1+j],Lcolor1 = midP[(K)*5*i+(K)*2+j], Lcolor2 = midP[(K)*5*i+(K)*3+j],Lcolor3 = midP[(K)*5*i+(K)*4+j];

	for (int k=0;k<spnum[i];k++)
	{
		Rlocation1 = midP[(K)*5*i+k],Rlocation2 = midP[(K)*5*i+(K)*1+k],Rcolor1 = midP[(K)*5*i+(K)*2+k], Rcolor2 = midP[(K)*5*i+(K)*3+k],Rcolor3 = midP[(K)*5*i+(K)*4+k];
		LDist=abs(Llocation1-Rlocation1)+abs(Llocation2-Rlocation2);
		CDist=sqrt((Lcolor1-Rcolor1)*(Lcolor1-Rcolor1)+(Lcolor3-Rcolor3)*(Lcolor3-Rcolor3)+(Lcolor2-Rcolor2)*(Lcolor2-Rcolor2));
		if (LDist<min(max(ring[i*K+k],*Par1),100.0))
		{
			weight1=exp(-CDist*(*Par));
			Tweight+=weight1;
			MSS=MSS+MSaliencyM[K*i+k]*weight1;
			CSS=CSS+CSaliencyM[K*i+k]*weight1;
		}
	}
if(i<N-1)
for (int k=0;k<spnum[i+1];k++)
	{
		Rlocation1 = midP[(K)*5*(i+1)+k],Rlocation2 = midP[(K)*5*(i+1)+(K)*1+k],Rcolor1 = midP[(K)*5*(i+1)+(K)*2+k], Rcolor2 = midP[(K)*5*(i+1)+(K)*3+k],Rcolor3 = midP[(K)*5*(i+1)+(K)*4+k];
		LDist=abs(Llocation1-Rlocation1)+abs(Llocation2-Rlocation2);
		CDist=sqrt((Lcolor1-Rcolor1)*(Lcolor1-Rcolor1)+(Lcolor3-Rcolor3)*(Lcolor3-Rcolor3)+(Lcolor2-Rcolor2)*(Lcolor2-Rcolor2));
		if (LDist<min(max(ring[(i+1)*K+k],*Par1),100.0))
		{
			weight1=exp(-CDist*(*Par));
			Tweight+=weight1;
			MSS=MSS+MSaliencyM[K*(i+1)+k]*weight1;
			CSS=CSS+CSaliencyM[K*(i+1)+k]*weight1;
		}
	}
if (i>0)
for (int k=0;k<spnum[i-1];k++)
	{
		Rlocation1 = midP[(K)*5*(i-1)+k],Rlocation2 = midP[(K)*5*(i-1)+(K)*1+k],Rcolor1 = midP[(K)*5*(i-1)+(K)*2+k], Rcolor2 = midP[(K)*5*(i-1)+(K)*3+k],Rcolor3 = midP[(K)*5*(i-1)+(K)*4+k];
		LDist=abs(Llocation1-Rlocation1)+abs(Llocation2-Rlocation2);
		CDist=sqrt((Lcolor1-Rcolor1)*(Lcolor1-Rcolor1)+(Lcolor3-Rcolor3)*(Lcolor3-Rcolor3)+(Lcolor2-Rcolor2)*(Lcolor2-Rcolor2));
		if (LDist<min(max(ring[(i-1)*K+k],*Par1),100.0))
		{
			weight1=exp(-CDist*(*Par));
			Tweight+=weight1;
			MSS=MSS+MSaliencyM[K*(i-1)+k]*weight1;
			CSS=CSS+CSaliencyM[K*(i-1)+k]*weight1;
		}
	}
	MSS=MSS/Tweight;
	CSS=CSS/Tweight;
	FinS[K*i+j]=MSS;
    FinC[K*i+j]=CSS;
	return;

}
void cudasmooth(double *FinS, double *FinC,double* midP,double * ring,double *MSaliencyM,double* CSaliencyM,double *K1,double *N1,double *Par,double *spnum,double *Par1)
{
	double * dev_FinS,*dev_FinC;
	double *dev_mid,*dev_ring,*dev_MSaliencyM,*dev_CSaliencyM;
    double *dev_K1,*dev_N1, *dev_Par,*dev_spnum,*dev_Par1;
    int K=(int)(*K1),N=(int)(*N1);

	hipMalloc((void **)&dev_mid, sizeof(double)* (K) * 5 * N);
	hipMalloc((void **)&dev_ring, sizeof(double)* K * N);
	hipMalloc((void **)&dev_CSaliencyM, sizeof(double)* K * N);
	hipMalloc((void **)&dev_MSaliencyM, sizeof(double)* K * N);
	hipMalloc((void **)&dev_FinS, sizeof(double)* K * N);
    hipMalloc((void **)&dev_FinC, sizeof(double)* K * N);
    hipMalloc((void **)&dev_K1, sizeof(double));
    hipMalloc((void **)&dev_N1, sizeof(double));
    hipMalloc((void **)&dev_Par, sizeof(double));
    hipMalloc((void **)&dev_Par1, sizeof(double));
    hipMalloc((void **)&dev_spnum, sizeof(double)*N);

	hipMemcpy(dev_K1, K1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_N1, N1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_mid, midP, sizeof(double)* (K) * 5 * N, hipMemcpyHostToDevice);
	hipMemcpy(dev_ring, ring, sizeof(double)* K*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_CSaliencyM, CSaliencyM, sizeof(double)* K*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_MSaliencyM, MSaliencyM, sizeof(double)* K*N, hipMemcpyHostToDevice);
	hipMemcpy(dev_Par, Par, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_Par1, Par1, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(dev_spnum, spnum, sizeof(double)*N, hipMemcpyHostToDevice);

	dim3 threads(K);
	dim3 grids(N);
	smooth_kernel << <grids, threads >> >(dev_FinS, dev_FinC, dev_mid,dev_ring,dev_MSaliencyM,dev_CSaliencyM,dev_K1,dev_N1,dev_Par,dev_spnum,dev_Par1);

	hipMemcpy(FinS, dev_FinS, sizeof(double)*K*N, hipMemcpyDeviceToHost);
    hipMemcpy(FinC, dev_FinC, sizeof(double)*K*N, hipMemcpyDeviceToHost);

	hipFree(dev_mid);
	hipFree(dev_FinS);
    hipFree(dev_FinC);
	hipFree(dev_CSaliencyM);
	hipFree(dev_MSaliencyM);
    hipFree(dev_ring);
    hipFree(dev_K1);
    hipFree(dev_N1);
    hipFree(dev_Par);
    hipFree(dev_spnum);
    hipFree(dev_Par1);
}	




